#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
// Add elements of two arrays
__global__ void add(int n, float* x, float* y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	y[index] = x[index] + y[index];
}

// Multiply two matrices
__global__ void matrix_mult(int n, float* x, float* y, float* c) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int res = 0;
	if (row < n && column < n) {
		for (int i = 0; i < n; i++)
			res += x[row * n + i] * y[i * n + column];
	}
	c[(row * n) + column] = res;
}

void matrix_main() {
	// N*N matrix
	int N = 30;
	float* x, * y, * c;

	hipMallocManaged(&x, N * N);
	hipMallocManaged(&y, N * N);
	hipMallocManaged(&c, N * N);

	// Randomize matrices
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			x[i * N + j] = rand() % 101;
			y[i * N + j] = rand() % 101;
		}
	}

	dim3 threadsPerBlock(N, N);
	matrix_mult<<<1, threadsPerBlock>>>(N, x, y, c);
	hipDeviceSynchronize();

	// Matrix multiply on CPU
	float* correctC = new float[N * N];
	float res;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			res = 0;
			for (int k = 0; k < N; k++) {
				res += x[i * N + k] * y[k * N + j];
			}
			correctC[i * N + j] = res;
		}
	}
	// Check for max errors
	double maxError = 0;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			maxError = fmax(maxError, fabs(c[(i * N) + j] - correctC[(i * N) + j]));
		}
	}
	std::cout << "Max error: " << maxError << std::endl;
	hipFree(x);
	hipFree(y);
	hipFree(c);
}

int main(void) {
	matrix_main();
	add_main();
}

void add_main()
{
	int N = 1 << 20; // 1M elements
	float* x, * y;
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));
	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	// Run kernel on 1M elements on the CPU
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	printf("%d", numBlocks);
	add<<<numBlocks, blockSize>>>(N, x, y);
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	std::cout << "Max error: " << maxError << std::endl;
	// Free memory
	hipFree(x);
	hipFree(y);
}